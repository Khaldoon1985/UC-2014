#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"
#include "reference.cpp"


__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code
   /*int id_x = blockIdx.x * blockDim.x + threadIdx.x ;
   int id_y = blockIdx.y * blockDim.y + threadIdx.y ;
   int absoulteId = id_y*numVals + id_x ;
     
   atomicAdd (&histo[vals[absoulteId]] , 1 );*/
    
   int id = blockIdx.x * blockDim.x + threadIdx.x ;
   atomicAdd (&histo[vals[id]] , 1 );


}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel

  //if you want to use/launch more than one kernel,
  //feel free

    //test algorithm
  /*int Nby2_ = numElems/2 ; 

  dim3 blocks (16,16);
  dim3 grid(Nby2_ / 16 , Nby2_/16) ; */

  //yourHisto <<<grid , blocks>>> (d_vals ,  d_histo, Nby2_ ) ;
    yourHisto <<<numElems/1024 , 1024>>> (d_vals ,  d_histo, numElems);

  /*delete[] h_vals;
  delete[] h_histo;
  delete[] your_histo;*/
}
