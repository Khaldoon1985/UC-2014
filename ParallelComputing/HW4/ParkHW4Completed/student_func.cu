#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */


__global__
void radixSort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems,
               unsigned int* const d_histogram,
               unsigned int* const d_scan,
               const unsigned int nBins
               )
{
    const unsigned int x = threadIdx.x;
    const unsigned int y = threadIdx.y;
    const unsigned int tIdx = x + y*blockDim.x;
    if (tIdx >= numElems)
        return;

    unsigned int* d_srcVals = d_inputVals;
    unsigned int* d_srcPos = d_inputPos;
    unsigned int* d_dstVals = d_outputVals;
    unsigned int* d_dstPos = d_outputPos;

    // iterate throught each bit
    for (unsigned int i = 0; i < 8 * sizeof(unsigned int); i += 1u) {
        const unsigned int mask = 1u << i;

        // initialize d_histogram, and d_scan
        if (tIdx < nBins) {
            d_histogram[tIdx] = 0u;
        }
        __syncthreads();

        // evaluate predicate, and compute histogram
        const unsigned int bin = (d_srcVals[tIdx % numElems] & mask) >> i;
        atomicAdd(&(d_histogram[bin % nBins]), 1u);
        __syncthreads();

        // compute the starting location of each bin with exclusive scan
        // This could be simple as
        if (tIdx == 0u) {
            d_scan[0] = 0u;
            d_scan[1] = d_histogram[0];
        }
        __syncthreads();

        // Move element to correct location
        const unsigned int offset = atomicAdd(&d_scan[bin % nBins], 1u);
        d_dstVals[offset % numElems] = d_srcVals[tIdx % numElems];
        d_dstPos[offset % numElems] = d_srcPos[tIdx% numElems];
        __syncthreads();

        // Swap the buffers
        unsigned int * tmp = d_srcVals;
        d_srcVals = d_dstVals;
        d_dstVals = tmp;

        tmp = d_srcPos;
        d_srcPos = d_dstPos;
        d_dstPos = tmp;
        __syncthreads();
    }
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
    // Step 1: Allocate memory
    const unsigned int nBins = 2u;
    unsigned int * d_histogram, * d_scan;
    checkCudaErrors( hipMalloc((void **)&d_histogram, nBins * sizeof(unsigned int)));
    checkCudaErrors( hipMalloc((void **)&d_scan, nBins * sizeof(unsigned int)));

    // Step 2: call radix sort
    std::cout << "Number of elements : " << numElems << std::endl;
    const dim3 blockSize(32, 32, 1);
    radixSort<<<1, blockSize>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems,
                                       d_histogram, d_scan, nBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // Step 3: release resources
    checkCudaErrors( hipFree(d_histogram));
    checkCudaErrors( hipFree(d_scan)); 
}
