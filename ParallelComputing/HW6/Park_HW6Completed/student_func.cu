#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */


#include "stdio.h"
#include "utils.h"
#include <thrust/host_vector.h>

__global__ void cal_mask( bool* d_mask, uchar4* d_source , int N){

    int n = threadIdx.x + blockDim.x * blockIdx.x;

    if (n >= N) return;
    
    if (d_source[n].x == 255 && d_source[n].y == 255 && d_source[n].z == 255) {
        d_mask[n] = 0; //outside the mask
    }
    
    else {
        d_mask[n] = 1; // inside the mask
    }
}

__global__ void isInside( int* d_inside, bool* d_mask , int N , int col_N){

    int n = threadIdx.x + blockDim.x * blockIdx.x;

    if (n >= N) {
        return;
    }

    if (d_mask[n] == true) {
    
        // is in mask, assume there are pixel around it.(never out of bound)
    
        if (d_mask[n-1] == true && d_mask[n+1] ==true && d_mask[n-col_N] == true && d_mask[n+col_N]== true) {
            // it's inside mask
            d_inside[n] = 1;
        }
        else {
            d_inside[n] = 0;
        }
    }
    else {
        d_inside[n] = 8;
    }
}

__global__ void separateChannels(float* red, float* blue, float* green, uchar4* d_source, int N) {

    int n = threadIdx.x + blockDim.x * blockIdx.x;

    if (n >= N) {
        return;
    }

    red[n]  = d_source[n].x;
    blue[n] = d_source[n].y;
    green[n]= d_source[n].z;
}

__global__ void jacobi( float* d_out, float* d_in , float* d_source, float* d_dest, int* d_inside, bool* d_mask , int N, int col) {
    
    int n = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (n >= N) {
        return;
    }

    if (d_mask[n] == 1) {
        if (d_inside[n] ==1 ) {
            float sum1 = 0.0f;
            float neighbor = 4.0f;
            float sum2 = 0.0f;

            if (d_inside[n-1] == 1) {
                sum1 += d_in[n-1];
                sum2 += d_source[n] - d_source[n-1];
            }
            
            else{
                // n-1 on the border
                sum1 += d_dest[n-1];
                sum2 += d_source[n] - d_source[n-1];
            }

            if (d_inside[n+1] == 1) {
                sum1 += d_in[n+1];
                sum2 += d_source[n] - d_source[n+1];
            }
            else{
                sum1 +=d_dest[n+1];
                sum2 += d_source[n] - d_source[n+1];
            }


            if (d_inside[n+col] == 1) {
                sum1 += d_in[n+col];
                sum2 += d_source[n] - d_source[n+col];
            }
            else{
                sum1 +=d_dest[n+col];
                sum2 += d_source[n] - d_source[n+col];
            }


            if (d_inside[n-col] == 1) {
                sum1 += d_in[n-col];
                sum2 += d_source[n] - d_source[n-col];
            }
           
            else{
                sum1 +=d_dest[n-col];
                sum2 += d_source[n] - d_source[n-col];
            }
            
            float newVal = (sum1 + sum2) / neighbor ;
            d_out[n] = min(255.f,max(0.f,newVal));

    
        }
    }
}

__global__ void storeBack( uchar4* d_dest, float* red, float* blue, float* green, int* d_inside , bool* d_mask, int N){

    int n = threadIdx.x + blockDim.x * blockIdx.x;

    if (n >= N) {
        return;
    }


    if (d_mask[n] == 1) {
    
        if (d_inside[n] == 1) {
        
            d_dest[n].x = char(red[n]  );
            d_dest[n].y = char(blue[n] );
            d_dest[n].z = char(green[n]);
        }
    }
}

void your_blend(
                //Input
                const uchar4* const h_sourceImg, const size_t numRowsSource, const size_t numColsSource, const uchar4* const h_destImg,
                //Output
                uchar4* const h_blendedImg) {

    printf("row: %lu, col: %lu\n", numRowsSource, numColsSource);

    int N = numRowsSource * numColsSource;
    bool* d_mask;
    int* d_inside;
    uchar4* d_source;
    uchar4* d_dest;
    float* d_source_red;
    float* d_source_blue;
    float* d_source_green;

    float *d_dest_red;
    float *d_dest_blue;
    float *d_dest_green;

    float *red;
    float *blue;
    float *green;

    float *red2;
    float *blue2;
    float *green2;

    checkCudaErrors(hipMalloc((void **)&d_mask   , N * sizeof(bool  ) )  );
    checkCudaErrors(hipMalloc((void **)&d_inside , N * sizeof(int   ) )  );
    checkCudaErrors(hipMalloc((void **)&d_source , N * sizeof(uchar4) )  );
    checkCudaErrors(hipMalloc((void **)&d_dest   , N * sizeof(uchar4) )  );
    checkCudaErrors(hipMalloc((void **)&d_source_red    , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&d_source_blue   , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&d_source_green  , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&d_dest_red    , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&d_dest_blue   , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&d_dest_green  , N * sizeof(float ) )  );

    checkCudaErrors(hipMalloc((void **)&red     , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&blue    , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&green   , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&red2     , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&blue2    , N * sizeof(float ) )  );
    checkCudaErrors(hipMalloc((void **)&green2   , N * sizeof(float ) )  );

    checkCudaErrors(hipMemcpy(d_source,h_sourceImg,N* sizeof(uchar4), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_dest  ,h_destImg  ,N* sizeof(uchar4), hipMemcpyHostToDevice));

    cal_mask<<<N/1024 + 1 , 1024 >>>(d_mask, d_source, N);

    hipDeviceSynchronize();
    
    isInside<<<N/1024 + 1 , 1024 >>>(d_inside, d_mask, N, numColsSource);

    separateChannels<<<N/1024 + 1, 1024>>>(d_source_red, d_source_blue, d_source_green, d_source, N);
    separateChannels<<<N/1024 + 1, 1024>>>(d_dest_red, d_dest_blue, d_dest_green, d_dest, N);
    separateChannels<<<N/1024 + 1, 1024>>>(red, blue, green, d_source, N);

    hipDeviceSynchronize();
    
    for (int i =0; i < 400; i++) {
        jacobi<<<N/1024 + 1, 1024>>>(red2, red, d_source_red, d_dest_red, d_inside, d_mask, N, numColsSource);
        jacobi<<<N/1024 + 1, 1024>>>(blue2, blue, d_source_blue, d_dest_blue,d_inside, d_mask, N, numColsSource);
        jacobi<<<N/1024 + 1, 1024>>>(green2, green, d_source_green, d_dest_green,d_inside, d_mask, N, numColsSource);
    
        hipDeviceSynchronize();
    
        jacobi<<<N/1024 + 1, 1024>>>(red, red2, d_source_red, d_dest_red, d_inside, d_mask, N, numColsSource);
        jacobi<<<N/1024 + 1, 1024>>>(blue, blue2, d_source_blue, d_dest_blue, d_inside, d_mask, N, numColsSource);
        jacobi<<<N/1024 + 1, 1024>>>(green, green2, d_source_green, d_dest_green, d_inside, d_mask, N, numColsSource);
    
        hipDeviceSynchronize();
    }



    storeBack<<<N/1024 + 1, 1024 >>>(d_dest, red, blue, green, d_inside, d_mask , N);

    hipDeviceSynchronize();
    
    checkCudaErrors(hipMemcpy(h_blendedImg,d_dest,N* sizeof(uchar4), hipMemcpyDeviceToHost));

    checkCudaErrors(hipGetLastError());
}